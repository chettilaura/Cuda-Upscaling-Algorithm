#include "hip/hip_runtime.h"
#include "imglib/img.h"
#include "cpulib/cpu.h"
#include "gpulib/gpu.cuh"
#include "standlib/stdCu.h"

int main(int argc, char **argv)
{
    // Go to help
    if (argc == 1 || (argc == 2 && (argv[1] == std::string("-h") || argv[1] == std::string("--help"))))
    {
        printf(
            "CUDA Upscale\n\n"
            "WARNING: This program is made only for educational purposes and is not intended to be used in production.\n\n"
            "Usage:\n\n"
            "    %s [Filtering Matrix generation's commands] inputFile.ppm cutOutCenterX cutOutCenterY cutOutWidth cutOutHeight zoomLevel [Matrix generation's parameters]\n\n"
            "  - Filtering Matrix generation's commands\n"
            "\t -c[v][f] --custom[v][f]: Generate a custom matrix from the file passed as an argument in the Matrix generation's parameters\n"
            "\t -g[v][f] --gauss[v][f]: Generate a gaussian matrix\n"
            "\t Optional: v character to allow verbose mode and print debug information\n"
            "\t Optional: f character to force the use of the global memory\n\n"
            "  - inputFile.ppm: A valid .ppm P6 input image\n"
            "  - cutOutCenterX: X coordinate of the center of the selection zone\n"
            "  - cutOutCenterY: Y coordinate of the center of the selection zone\n"
            "  - cutOutWidth: Length of the side of the selection\n\n"
            "  - cutOutHeight: Length of the side of the selection\n\n"
            "  - zoomLevel: Zoom level of the output image, must be a INT value from 1 to 32\n"
            "               If 1 is inserted, only the convolution will be performed\n\n"
            "  - Matrix generation's parameters\n"
            "\t GaussLength: must be an odd value from 3 to 15 sides included\n"
            "\t GaussSigma: must be a value from 0.5 to 5 sides included\n"
            "\t InputKernelFile.txt: formatted as such\n\n"
            "\t\t\tmatrixSide'sLength (must be odd)\n"
            "\t\t\tFirstElement SecondElement ...\n"
            "\t\t\tRowElement ...\n"
            "\t\t\t...\n",
            argv[0]);
    }

    if (argc < 9 || argc > 10)
    {
        printf("Wrong command line input. Use -h or --help for more information\n");
        return -1;
    }

    // Check if verbose mode is enabled
    std::string arg1 = argv[1];
    bool verbose = arg1.find('v') != std::string::npos;
    bool forceGlobal = arg1.find('f') != std::string::npos;

    // Check GPU
    int nDevices;
    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(err));
        return -1;
    }
    if (nDevices == 0)
    {
        printf("No CUDA device found\n");
        return -1;
    }

    if (verbose)
        printf("Number of CUDA devices: %d\n", nDevices);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // Mask length
    char maskDim = 0;

    // Check mode and load kernel
    if (verbose)
        printf("Loading kernel...\n");

    // Custom Kernel from file
    if (argc == 9 && ((arg1.rfind("-c", 0) == 0) || (arg1.rfind("--custom", 0) == 0)))
    {
        FILE *kernelFile = fopen(argv[8], "r");
        if (kernelFile == NULL)
        {
            printf("Error opening file %s\n", argv[8]);
            return -1;
        }

        // Read file
        char buff[120];
        fgets(buff, 100, kernelFile);
        maskDim = (char)strtol(buff, NULL, 10);
        if (maskDim < 3 || maskDim > 15 || maskDim % 2 == 0)
        {
            printf("Wrong mask dimension. Use -h or --help for more information\n");
            return -1;
        }

        // Allocate memory
        float *kernel = (float *)malloc(maskDim * maskDim * sizeof(float));
        for (int i = 0; i < maskDim; i++)
        {
            fgets(buff, 120, kernelFile);
            if (sscanf(buff, "%f %f %f %f %f %f %f %f %f", &kernel[i * maskDim], &kernel[i * maskDim + 1], &kernel[i * maskDim + 2], &kernel[i * maskDim + 3], &kernel[i * maskDim + 4], &kernel[i * maskDim + 5], &kernel[i * maskDim + 6], &kernel[i * maskDim + 7], &kernel[i * maskDim + 8]) != maskDim)
            {
                printf("Wrong input. Use -h or --help for more information\n");
                return -1;
            }
            else if (verbose)
                printf("%f %f %f %f %f %f %f %f %f \n", kernel[i * maskDim], kernel[i * maskDim + 1], kernel[i * maskDim + 2], kernel[i * maskDim + 3], kernel[i * maskDim + 4], kernel[i * maskDim + 5], kernel[i * maskDim + 6], kernel[i * maskDim + 7], kernel[i * maskDim + 8]);
        }
        fclose(kernelFile);

        // Copy to device
        loadKernel(kernel, maskDim);
        free(kernel);
    }
    // Gaussian kernel
    else if (argc == 10 && ((arg1.rfind("-g", 0) == 0) || (arg1.rfind("--gauss", 0) == 0)))
    {
        int gaussLength = (int)strtol(argv[8], NULL, 10);
        float gaussSigma = (float)strtof(argv[9], NULL);
        if (gaussLength < 3 || gaussLength > 15 || gaussLength % 2 == 0 || gaussSigma < 0.5 || gaussSigma > 5)
        {
            printf("Wrong Gaussian values:\nACCEPTED VALUES:\n\t 3 <= gaussLength (must be odd) <= 15\n\t 0.5 <= gaussSigma <= 5\nAborting...\n");
            return -1;
        }
        float *gaussKernel = (float *)malloc(gaussLength * gaussLength * sizeof(float));
        gaussianKernelCPU(gaussLength, gaussSigma, gaussKernel);
        if (verbose)
        {
            printf("\nGaussLength: %d\nGaussSigma: %f\n", gaussLength, gaussSigma);
            printf("\nGaussian kernel:\n");
            for (int i = 0; i < gaussLength; i++)
            {
                for (int j = 0; j < gaussLength; j++)
                {
                    printf("%f ", gaussKernel[i * gaussLength + j]);
                }
                printf("\n");
            }
            printf("\n");
        }
        maskDim = gaussLength;
        loadKernel(gaussKernel, maskDim);
        free(gaussKernel);
    }
    // Display Error
    else
    {
        printf("Wrong command line input. Use -h or --help for more information\n");
        printf("arg1: %s, argc: %d\n", arg1.c_str(), argc);
        return -1;
    }
    if (verbose)
        printf("Kernel loaded\n"
               "Mask dimension: %d\n"
               "Proceeding with checks for scaling...\n",
               maskDim);

    const int cutOutCenterX = (int)strtol(argv[3], NULL, 10); // X coordinate of the center of the selection zone
    const int cutOutCenterY = (int)strtol(argv[4], NULL, 10); // Y coordinate of the center of the selection zone
    const int cutOutWidth = (int)strtol(argv[5], NULL, 10);   // Length of the side of the selection mask
    const int cutOutHeight = (int)strtol(argv[6], NULL, 10);  // Length of the side of the selection mask
    const int zoomLevel = (int)strtol(argv[7], NULL, 10);     // Zoom level

    // check cutOutWidth is even
    if (cutOutWidth % 2 != 0)
    {
        printf("Error: cutOutWidth must be even\n");
        return -1;
    }

    // check cutOutHeight is even
    if (cutOutHeight % 2 != 0)
    {
        printf("Error: cutOutHeight must be even\n");
        return -1;
    }

    if (zoomLevel < 1 || zoomLevel > 32)
    {
        printf("Error: zoomLevel must be between 1 and 32\n");
        return -1;
    }

    if (verbose)
        printf("cutOutCenterX: %d, cutOutCenterY: %d, cutOutWidth: %d, cutOutHeight: %d, zoomLevel: %d\n", cutOutCenterX, cutOutCenterY, cutOutWidth, cutOutHeight, zoomLevel);

    // Check input file ends with .ppm
    if (std::string(argv[2]).size() < 4 || std::string(argv[2]).substr(std::string(argv[2]).size() - 4) != ".ppm")
    {
        printf("Error: input file must be a .ppm file\n");
        return -1;
    }
    RGBImage *img = readPPM(argv[2]);

    // Y boundaries check and mask check
    const int pointY = cutOutCenterY - cutOutHeight / 2;
    if (cutOutCenterY > img->height || cutOutCenterY < 0)
    {
        printf("Error: cutOutCenterY outside image boundaries\n");
        return -1;
    }
    if ((cutOutCenterY + cutOutHeight / 2) > img->height - 1 || pointY < 1)
    {
        printf("Error: Y mask outside image boundaries\n");
        return -1;
    }

    // X boundaries check and mask check
    const int pointX = cutOutCenterX - cutOutWidth / 2;
    if (cutOutCenterX > img->width || cutOutCenterX < 0)
    {
        printf("Error: cutOutCenterX outside image boundaries\n");
        return -1;
    }
    if ((cutOutCenterX + cutOutWidth / 2) > img->width - 1 || pointX < 1)
    {
        printf("Error: X mask outside image boundaries\n");
        return -1;
    }

    const int widthImgIn = img->width;
    const int heightImgIn = img->height;
    const int widthImgOut = cutOutWidth * zoomLevel;
    const int heightImgOut = cutOutHeight * zoomLevel;
    const int outPx = widthImgOut * heightImgOut * 3;

    unsigned char *d_start, *d_out;
    hipMalloc((void **)&d_start, img->height * img->width * 3 * sizeof(char));
    hipMemcpy(d_start, img->data, img->height * img->width * 3 * sizeof(char), hipMemcpyHostToDevice);
    destroyPPM(img);
    hipMalloc((void **)&d_out, outPx * sizeof(char));

    if (verbose)
        printf("Image loaded\n"
               "Image width: %d px\n"
               "Image height: %d px\n"
               "Image size: %d bytes\n"
               "Output width: %d px\n"
               "Output height: %d px\n"
               "Output size: %d bytes\n",
               widthImgIn, img->height, img->height * img->width * 3, widthImgOut, heightImgOut, outPx);

    int widthTile = ((int)sqrt(prop.maxThreadsPerBlock) - (maskDim - 1));
    int heightTile = widthTile;
    if (checkTiling(widthImgOut, heightImgOut, &widthTile, &heightTile) && !forceGlobal)
    // TRUE: Tiling approach doable
    {
        // Number of threads per block
        dim3 usedThreads = dim3(widthTile + maskDim - 1, heightTile + maskDim - 1, 1);
        // Number of blocks
        dim3 usedBlocks = dim3(widthImgOut / widthTile, heightImgOut / heightTile, 3);
        if(usedBlocks.x > prop.maxGridSize[0] || usedBlocks.y > prop.maxGridSize[1] || usedBlocks.z > prop.maxGridSize[2])
        {
            printf("Error: Blocks overflow\n");
            return -1;
        }
        
        // Bytes of shared memory per block
        int sharedMemSize = (widthTile + maskDim - 1) * (heightTile + maskDim - 1) * sizeof(char);
        if (sharedMemSize > prop.sharedMemPerBlock)
        {
            printf("Error: Shared memory overflow\n");
            return -1;
        }

        if (verbose)
            printf("Tiling approach executing...\n"
                   "Threads per block: %d, %d, %d\n"
                   "Blocks: %d, %d, %d\n"
                   "Shared memory size: %d bytes\n"
                   "Launching kernel...\n"
                   "Parameters:\n"
                   "",
                   usedThreads.x, usedThreads.y, usedThreads.z, usedBlocks.x, usedBlocks.y, usedBlocks.z, sharedMemSize);

        tilingCudaUpscaling<<<usedBlocks, usedThreads, sharedMemSize>>>(d_start, d_out, widthImgIn, heightImgIn, widthImgOut, heightImgOut, widthTile, heightTile, maskDim, (pointX - (maskDim / 2 / zoomLevel)), (pointY - (maskDim / 2 / zoomLevel)), zoomLevel);
    }
    else
    // FALSE: Global memory approach is used
    {
        // Number of threads per block
        dim3 usedThreads = (outPx > prop.maxThreadsPerBlock) ? prop.maxThreadsPerBlock : outPx;
        // Number of blocks
        dim3 usedBlocks = (outPx / prop.maxThreadsPerBlock) + 1;
        if (usedBlocks.x > prop.maxGridSize[0])
        {
            printf("%s\n", hipGetErrorString(err));
            return -1;
        }

        if (verbose)
            printf("Global memory approach executing...\n"
                   "Threads per block: %d\n"
                   "Blocks: %d\n"
                   "Launching kernel...\n",
                   usedThreads.x, usedBlocks.x);

        globalCudaUpscaling<<<usedBlocks, usedThreads>>>(d_start, d_out, widthImgIn, heightImgIn, widthImgOut, heightImgOut, maskDim, (pointX - (maskDim / 2 / zoomLevel)), (pointY - (maskDim / 2 / zoomLevel)), zoomLevel);
    }
    hipDeviceSynchronize();

    // Check for errors
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(err));
        return -1;
    }

    // Copy result back to host
    RGBImage *out = createPPM(widthImgOut, heightImgOut);
    hipMemcpy(out->data, d_out, outPx * sizeof(char), hipMemcpyDeviceToHost);

    hipFree(d_start);
    hipFree(d_out);

    // Write output file
    writePPM("output.ppm", out);
    destroyPPM(out);

    if (verbose)
        printf("END OF THE PROGRAM\n\n");

    return 0;
}