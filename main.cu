#include "hip/hip_runtime.h"
#include "imglib/img.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define N 9

signed char sharpness[N] = {0, -1, 0, -1, 4, -1, 0, -1, 0};
__constant__ float mask[N];

__global__ void convGPU(char *input, char *output, const int dim)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= dim*dim)
        return;

    output[idx] = input[idx] * mask [0] + input[idx + 1] * mask [1] + input[idx + 2] * mask [2] + input[idx + dim] * mask [3] + input[idx + dim + 1] * mask [4] + input[idx + dim + 2] * mask [5] + input[idx + dim*2] * mask [6] + input[idx + dim*2 + 1] * mask [7] + input[idx + dim*2 + 2] * mask [8];
}

void zero_order_zooming(int *img, int *zoomed_out, int dimZoomX, int dimZoomY, int x, int y, int width, int height)
{

    int x_range_max;
    int y_range_max;

    int zoomed = malloc(dimZoomX * dimZoomY * sizeof(int));
    int zoomed_out = malloc(width * height * sizeof(int));

    if (x < 0 || y < 0 || x > width || y > height)
    {
        printf("Errore: coordinate fuori dai bordi dell'immagine");
        return;
    }

    if (width - x < x)
    {
        x_range_max = width - x;
    }
    else
    {
        x_range_max = x;
    }

    if (height - y < y)
    {
        y_range_max = height - y;
    }
    else
    {
        y_range_max = y;
    }

    if (x_range_max < dimZoomX || y_range_max < dimZoomY)
    {
        printf("Errore: dimensione della maschera fuori dai bordi dell'immagine");
        return;
    }

    for (int i = 0; i < dimZoomY; i++)
        for (int j = 0; j < dimZoomX; j++)
            zoomed[i * dimZoomX + j] = img[x + j + (y + i) * width];

    float stuffing_bits_x = width / dimZoomX;
    float stuffing_bits_y = height / dimZoomY;

    int stuffing_x = (int)stuffing_bits_x;
    int stuffing_y = (int)stuffing_bits_y;

    int x_float_stuff = (int)100 / (stuffing_bits_x * 100 - stuffing_x * 100);
    int y_float_stuff = (int)100 / (stuffing_bits_y * 100 - stuffing_y * 100);

    int x_float_stuff_counter = 0;
    int y_float_stuff_counter = 0;

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            zoomed_out[i * width + j] = zoomed[(i / stuffing_y) * dimZoomX + (j / stuffing_x)];
            if (x_float_stuff_counter == x_float_stuff)
            {
                zoomed_out[i * width + j] = zoomed[(i / stuffing_y) * dimZoomX + (j / stuffing_x) + 1];
                x_float_stuff_counter = 0;
            }
            if (y_float_stuff_counter == y_float_stuff)
            {
                zoomed_out[i * width + j] = zoomed[((i / stuffing_y) + 1) * dimZoomX + (j / stuffing_x)];
                y_float_stuff_counter = 0;
            }
            x_float_stuff_counter++;
            y_float_stuff_counter++;
        }
    }

}

__global__ void scaleGPU(char *input, char *output, const int dim, const int dimSmall)
{    
}

int main(int argc, char **argv)
{
    // Filter Setup
    switch (argc)
    {
    // Help
    case 2:
        if (argv[1] == std::string("-h") || argv[1] == std::string("--help"))
        {
            printf("Usage: %s inputFile.ppm DimX DimY dimZoom mode [GaussLength GaussSigma]\n\nWhere:\n\tDimX: X coordinate of the center of the selection zone\n\tDimY: Y coordinate of the center of the selection zone\n\tdimZoom: Length of the side of the selection mask which has a square shape\n\tmode: 0 for Sharpness filter, 1 for Gaussian*, 2 for custom 3x3 Kernel*\n*Input required from the user\n", argv[0]);
            return 0;
        }
        else
        {
            printf("Wrong command line input. Use -h or --help for more information\n");
            return -1;
        }
        break;

    // Sharpness or Custom Kernel
    case 6:{
        int mode = (int)strtol(argv[5], NULL, 10);
        if (mode == 0)
            hipMemcpyToSymbol(HIP_SYMBOL(mask), sharpness, N * sizeof(char));
        else if (mode == 2)
        {
            printf("Insert the 3x3 kernel values, from left to right, from top to bottom to a single line where each value is separated by a space\n");
            const char M = N * 4;
            char buff[M];
            fgets(buff, M, stdin);

            // Check input
            int bLength = strlen(buff);
            if (bLength < 18) // 9 values, 8 spaces
            {
                printf("Wrong input. Use -h or --help for more information\n");
                return -1;
            }
            float *kernel = (float *)malloc(N * sizeof(float));
            if (sscanf(buff, "%f %f %f %f %f %f %f %f %f", &kernel[0], &kernel[1], &kernel[2], &kernel[3], &kernel[4], &kernel[5], &kernel[6], &kernel[7], &kernel[8]) != 9)
            {
                printf("Wrong input. Use -h or --help for more information\n");
                return -1;
            }
            hipMemcpyToSymbol(HIP_SYMBOL(mask), kernel, N * sizeof(float));
            free(kernel);
        }
        else
        {
            printf("Wrong command line input. Use -h or --help for more information\n");
            return -1;
        }
    }
        break;

    // Gaussian
    case 8:{
        int mode = (int)strtol(argv[5], NULL, 10);
        if (mode != 1)
        {
            printf("Wrong command line input. Do not input gaussian data for non-gaussian matrices. Use -h or --help for more information\n");
            return -1;
        }
        int gaussLength = (int)strtol(argv[6], NULL, 10);
        float gaussSigma = (float)strtof(argv[7], NULL);
        if (gaussLength < 3 || gaussLength > 15 || gaussSigma < 0.5 || gaussSigma > 5)
        {
            printf("Wrong Gaussian values:\nACCEPTED VALUES:\n\t 3 <= gaussLength <= 15\n\t 0.5 <= gaussSigma <= 5\nAborting...\n");
            return -1;
        }
        float *gaussKernel = (float *)malloc(N * sizeof(float));
        // TODO: implementare gaussianKernel
        // gaussKernel = gaussianKernel(gaussLength, gaussSigma);
        hipMemcpyToSymbol(HIP_SYMBOL(mask), gaussKernel, N * sizeof(float));
        free(gaussKernel);
    }
        break;
    
    default:
        printf("Wrong command line input. Use -h or --help for more information\n");
        return -1;
    }

    dimX = (int) strtol(argv[2], NULL, 10);
    dimY = (int) strtol(argv[3], NULL, 10);
    dimZoom = (int) strtol(argv[4], NULL, 10);

    printf("DimX: %d, DimY: %d, dimZoom: %d\n", dimX, dimY, dimZoom);


    GrayImage *img = readPGM(argv[1]);
    if (img == NULL)
    {
        printf("Errore: dimZoom deve essere pari\n");
        return -1;
    }

    printf("DimX: %d, DimY: %d, dimZoom: %d\n", dimX, dimY, dimZoom);

    RGBImage *img = readPPM(argv[1]);

    // Check per Y
    if (dimY > img->height || dimY < 0)
    {
        printf("Errore: Y fuori dai bordi dell'immagine");
        return -1;
    }
    {
        const int boardYup = dimY - dimZoom / 2;
        const int boardYdown = dimY + dimZoom / 2;
        if (boardYdown > img->height - 1 || boardYup < 1)
        {
            printf("Errore: Maschera Y fuori dai bordi dell'immagine");
            return -1;
        }
    }

    // Check per X
    if (dimX > img->width || dimX < 0)
    {
        printf("Errore: X fuori dai bordi dell'immagine");
        return -1;
    }
    {
        const int boardXup = dimX + dimZoom / 2;
        const int boardXdown = dimX - dimZoom / 2;
        if (boardXup > img->width - 1 || boardXdown < 1)
        {
            printf("Errore: Maschera X fuori dai bordi dell'immagine");
            return -1;
        }
    }

    // Selezione
    // What is the order between scaling and convolution?
    
    const int inConvDim = dimZoom + 2;
    const int outScaleDim = (img->width >= img->height) ? img->width : img->height;    
    const int pxCount = outScaleDim * outScaleDim * 3;
    RGBImage *imgConvWorked = createPPM(inConvDim, inConvDim);
    RGBImage *imgScaled = createPPM(outScaleDim, outScaleDim);
    
    unsigned char *startingMatrix = (unsigned char *)malloc(inConvDim * inConvDim * 3 * sizeof(unsigned char));    
    const int pointX = dimX - dimZoom / 2;
    const int pointY = dimY - dimZoom / 2;

    for (int i = 0; i < dimZoom; i++)
        for (int j = 0; j < dimZoom; j++)
        {
            startingMatrix[(i + 1) * inConvDim * 3 + (j + 1) * 3] = img->data[(pointX + j) * 3 + (pointY + i) * img->width * 3];
            startingMatrix[(i + 1) * inConvDim * 3 + (j + 1) * 3 + 1] = img->data[(pointX + j) * 3 + (pointY + i) * img->width * 3 + 1];
            startingMatrix[(i + 1) * inConvDim * 3 + (j + 1) * 3 + 2] = img->data[(pointX + j) * 3 + (pointY + i) * img->width * 3 + 2];
        }

    destroyPPM(img);

    char *d_start, *d_Scale, *d_Conv;    

    hipMalloc((void **)&d_start, inConvDim * inConvDim * 3 * sizeof(char));
    hipMalloc((void **)&d_Scale, outScaleDim * outScaleDim * 3 * sizeof(char));
    hipMalloc((void **)&d_Conv, dimZoom * dimZoom * 3 * sizeof(char));
    hipMemcpy(d_start, startingMatrix, inConvDim * inConvDim * 3 * sizeof(char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    free(startingMatrix);

    // Check GPU
    int nDevices;
    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) {
        printf("%s\n", hipGetErrorString(err));
        return -1;
    }


    
    int *d_imgCenter, *d_imgLeft, *d_imgRight, *d_imgUp, *d_imgDown, *d_imgConv;
    hipMalloc((void **)&d_imgCenter, dimZoom * dimZoom * sizeof(int));
    hipMalloc((void **)&d_imgLeft, dimZoom * dimZoom * sizeof(int));
    hipMalloc((void **)&d_imgRight, dimZoom * dimZoom * sizeof(int));
    hipMalloc((void **)&d_imgUp, dimZoom * dimZoom * sizeof(int));
    hipMalloc((void **)&d_imgDown, dimZoom * dimZoom * sizeof(int));
    hipMalloc((void **)&d_imgConv, dimZoom * dimZoom * sizeof(int));


    // Convoluzione
    GrayImage *imgConv = createPGM(dimZoom, dimZoom);
    int thread= dimZoom*dimZoom;
    superSampler<<<1, 1>>>(d_imgCenter, d_imgLeft, d_imgRight, d_imgUp, d_imgDown, d_imgConv, dimZoom);
    hipDeviceSynchronize();


    // Scale
    const int blockCeilingScale = (pxCount / maxThreads) + 1;
    scaleGPU<<<blockCeilingScale, maxThreads>>>(d_Conv, d_Scale, outScaleDim, dimZoom);    
    hipDeviceSynchronize();

    //hipMemcpy(imgFinalWorked->data, d_endScale, pxCount * sizeof(char), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(d_start);
    hipFree(d_Conv);
    hipFree(d_Scale);
    printf("well done");

    // Stampa
    //writePPM("output.ppm", imgFinalWorked);
    destroyPPM(imgScaled);
    destroyPPM(imgConvWorked);

    return 0;
}